
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#define MAX 32

 

__global__ void toggle_case(char *x, char *y)
{
int i=threadIdx.x;
if(x[i] >= 'a' && x[i] <='z')
y[i] = x[i]-32;
else
if(x[i] >= 'A' && x[i] <='Z')
y[i] = x[i]+32;
else
{
y[i] = x[i];
}
}

 

int main()
{
int n;
hipEvent_t start, stop;
float time;
char A[MAX],B[MAX],*d,*e;
printf("Enter String to be toggled: ");
scanf("%s",A);
n = strlen(A)+1;
hipEventCreate(&start);
hipEventCreate(&stop);
hipMalloc((void **)&d,n*sizeof(char));
hipMalloc((void **)&e,n*sizeof(char));
hipMemcpy(d,A,n*sizeof(char),hipMemcpyHostToDevice);
hipEventRecord(start, 0);
toggle_case<<<1,n>>>(d,e);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipMemcpy(B,e,n*sizeof(char),hipMemcpyDeviceToHost);
printf("The toggled case String is %s", B);
printf("\n");
hipFree(d);
hipFree(e);
hipEventElapsedTime(&time, start, stop);
printf ("Time for the kernel: %f ms\n", time);
return 0;
}
